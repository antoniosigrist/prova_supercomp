#include "hip/hip_runtime.h"
#include <iostream>
#include <unistd.h>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#define size 21  // Tamanho da matrix

// Exibe os pontos na tela
void print(bool grid[][size]){
  std::cout << "\n\n\n\n\n";
  for(unsigned int i = 1; i < size-1; i++) {
    for(unsigned int j = 1; j < size-1; j++)
      std::cout << (grid[i][j]?"#":"_");
    std::cout << std::endl;
  }
}

// Calcula a simulacao
__global__ bool jogo(bool grid[][size]){
  bool isAlive = false;
  bool grid_tmp[size][size] = {};
  for(unsigned int i=0; i < size; i++)
    for(unsigned int j=0; j < size; j++)
      grid_tmp[i][j] = grid[i][j]; // copia os daods do grid
  for(unsigned int i = 1; i < size-1; i++)
    for(unsigned int j = 1; j < size-1; j++) {
      unsigned int count = 0;
      if(grid[i][j]) isAlive = true;
      for(int k = -1; k <= 1; k++) 
        for(int l = -1; l <= 1; l++)
          if(k != 0 || l != 0)
            if(grid_tmp[i+k][j+l])
              ++count;
      if(count < 2 || count > 3) grid[i][j] = false;
      else if(count == 3) grid[i][j] = true;
    }

}

int main(){
  bool grid[size][size] = {}; // dados iniciais
  int parada = 0;
  int* dEnv;

  hipMalloc((void**) &dEnv, size * size * sizeof(bool));
  hipMemcpy(dEnv, grid, size * size * sizeof(bool), hipMemcpyHostToDevice);

  dim3 golThreads(size, size);

  grid[ 5][ 7] = true;
  grid[ 6][ 8] = true;
  grid[ 8][ 8] = true;
  grid[ 6][ 9] = true;
  grid[ 8][10] = true;
  grid[ 9][10] = true;
  grid[ 8][11] = true;
  grid[10][11] = true;
  grid[10][12] = true;

  while (parada<100) { // loop enquanto algo vivo

    system("clear");

    jogo<<<1,golThreads>>>(dEnv)

    hipMemcpy(grid, dEnv, size * size * sizeof(bool), hipMemcpyDeviceToHost);

    print(grid);

    usleep(100000);  // pausa para poder exibir no terminal

    parada++;
  } 
}
