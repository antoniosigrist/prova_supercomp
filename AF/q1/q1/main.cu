
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>


#define size 21

using namespace std;

__global__ void jogo(bool* env) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  // mapeaia as bordas da posição analisada

  int wrapNorth = ((size + y - 1) % size) * size;
  int wrapSouth = ((size + y + 1) % size) * size;

  int wrapEast = (size + x + 1) % size;
  int wrapWest = (size + x - 1) % size;

  // conta quantos existem
  int count = 0;

  if (env[y * size + wrapEast]) count++;
  if (env[y * size + wrapWest]) count++;
  if (env[wrapNorth + wrapEast]) count ++;
  if (env[wrapNorth + wrapWest]) count++;
  if (env[wrapSouth + wrapEast]) count++;
  if (env[wrapSouth + wrapWest]) count++;
  if (env[wrapNorth + x]) count++;
  if (env[wrapSouth + x]) count++;


  __syncthreads(); //garante que as threads estejam sincronizadas para realizar o calculo de vizinhos vivos


  if(count < 2 || count > 3)
    env[y * size + x] = false;

  if(count == 3)
    env[y * size + x] = true;
}

void print(bool* env) {
  for(int i = 0; i < size * size; i++) {

    cout << (env[i] ? '#' : ' ');

    if (!(i % size)) cout << endl;
  }
}

int main(){

  int parada = 0;

  bool env[size * size]; //linearizei o vetor


  env[ 5*size + 7] = true;
  env[ 6*size + 8] = true;
  env[ 8*size +8] = true;
  env[ 6*size +6] = true;
  env[ 8*size +10] = true;
  env[ 9*size +10] = true;
  env[ 8*size +11] = true;
  env[10*size +11] = true;
  env[10*size +12] = true;

  bool* dEnv;

  hipMalloc((void**) &dEnv, size * size * sizeof(bool)); //aloca vetor em cuda
  hipMemcpy(dEnv, env, size * size * sizeof(bool), hipMemcpyHostToDevice); //copia o vetor para cuda

  dim3 golThreads(size, size); //define tamanho das threads

  while (parada < 100) { //define o numero de parada para 100
    system("clear");
    jogo<<<1, golThreads>>>(dEnv); //chamada do kernel
    hipMemcpy(env, dEnv, size * size * sizeof(bool), hipMemcpyDeviceToHost); //copia valor do vetor de volta a cpu
    print(env);

    usleep(100000);

    parada++;
  }
}