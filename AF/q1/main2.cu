#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define size 21

using namespace std;

__global__ void gpuNext(int* env) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  int wrapNorth = ((size + y - 1) % size) * size;
  int wrapSouth = ((size + y + 1) % size) * size;

  int wrapEast = (size + x + 1) % size;
  int wrapWest = (size + x - 1) % size;

  int neighbours =
    env[y * size + wrapEast] + // EAST + MIDDLE
    env[y * size + wrapWest] + // WEST + MIDDLE

    env[wrapNorth + wrapEast] + // EAST + NORTH
    env[wrapNorth + wrapWest] + // WEST + NORTH

    env[wrapSouth + wrapEast] + // EAST + SOUTH
    env[wrapSouth + wrapWest] + // WEST + SOUTH

    env[wrapNorth + x] + // MIDDLE + TOP
    env[wrapSouth + x]; // MIDDLE + BOTTOM

  __syncthreads();

  if(neighbours < 2 || neighbours > 3)
    env[y * size + x] = 0;

  if(neighbours == 3)
    env[y * size + x] = 1;
}

void print(int* env) {
  for(int i = 0; i < size * size; i++) {
    cout << (env[i] ? '#' : ' ');

    if (!(i % size)) cout << endl;
  }
}

int main(){
  bool env[size * size];

  // srand(time(NULL));

  // for (int i = 0; i < size * size; i++) {
  //   env[i] = rand() % 2 == 0;
  // }

  env[ 5*size + 7] = true;
  env[ 6*size + 8] = true;
  env[ 8*size +8] = true;
  env[ 6*size +6] = true;
  env[ 8*size +10] = true;
  env[ 9*size +10] = true;
  env[ 8*size +11] = true;
  env[10*size +11] = true;
  env[10*size +12] = true;

  bool* dEnv;

  hipMalloc((void**) &dEnv, size * size * sizeof(bool));
  hipMemcpy(dEnv, env, size * size * sizeof(bool), hipMemcpyHostToDevice);

  dim3 golThreads(size, size);

  while (true) {
    system("clear");
    gpuNext<<<1, golThreads>>>(dEnv);
    hipMemcpy(env, dEnv, size * size * sizeof(bool), hipMemcpyDeviceToHost);
    print(env);
    system("sleep .1");
  }
}