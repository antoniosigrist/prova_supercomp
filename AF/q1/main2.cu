
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define HEIGHT 32
#define WIDTH 32

using namespace std;

__global__ void gpuNext(int* env) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  int wrapNorth = ((HEIGHT + y - 1) % HEIGHT) * WIDTH;
  int wrapSouth = ((HEIGHT + y + 1) % HEIGHT) * WIDTH;

  int wrapEast = (WIDTH + x + 1) % WIDTH;
  int wrapWest = (WIDTH + x - 1) % WIDTH;

  int neighbours =
    env[y * WIDTH + wrapEast] + // EAST + MIDDLE
    env[y * WIDTH + wrapWest] + // WEST + MIDDLE

    env[wrapNorth + wrapEast] + // EAST + NORTH
    env[wrapNorth + wrapWest] + // WEST + NORTH

    env[wrapSouth + wrapEast] + // EAST + SOUTH
    env[wrapSouth + wrapWest] + // WEST + SOUTH

    env[wrapNorth + x] + // MIDDLE + TOP
    env[wrapSouth + x]; // MIDDLE + BOTTOM

  __syncthreads();

  if(neighbours < 2 || neighbours > 3)
    env[y * WIDTH + x] = 0;

  if(neighbours == 3)
    env[y * WIDTH + x] = 1;
}

void print(int* env) {
  for(int i = 0; i < WIDTH * HEIGHT; i++) {
    cout << (env[i] ? '#' : ' ');

    if (!(i % WIDTH)) cout << endl;
  }
}

int main(){
  int env[WIDTH * HEIGHT];

  srand(time(NULL));

  for (int i = 0; i < WIDTH * HEIGHT; i++) {
    env[i] = rand() % 2;
  }

  int* dEnv;

  hipMalloc((void**) &dEnv, WIDTH * HEIGHT * sizeof(int));
  hipMemcpy(dEnv, env, WIDTH * HEIGHT * sizeof(int), hipMemcpyHostToDevice);

  dim3 golThreads(WIDTH, HEIGHT);

  while (true) {
    system("clear");
    gpuNext<<<1, golThreads>>>(dEnv);
    hipMemcpy(env, dEnv, WIDTH * HEIGHT * sizeof(int), hipMemcpyDeviceToHost);
    print(env);
    system("sleep .1");
  }
}