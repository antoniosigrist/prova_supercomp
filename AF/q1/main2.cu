
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>


#define size 21

using namespace std;

__global__ void jogo(bool* env) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  // mapeaia as bordas da posição analisada

  int wrapNorth = ((size + y - 1) % size) * size;
  int wrapSouth = ((size + y + 1) % size) * size;

  int wrapEast = (size + x + 1) % size;
  int wrapWest = (size + x - 1) % size;

  // conta quantos existem
  int count = 0;

  if (env[y * size + wrapEast]) count++;
  if (env[y * size + wrapWest]) count++;
  if (env[wrapNorth + wrapEast]) count ++;
  if (env[wrapNorth + wrapWest]) count++;
  if (env[wrapSouth + wrapEast]) count++;
  if (env[wrapSouth + wrapWest]) count++;
  if (env[wrapNorth + x]) count++;
  if (env[wrapSouth + x]) count++;



  // int neighbours =
  //   env[y * size + wrapEast] + // EAST + MIDDLE
  //   env[y * size + wrapWest] + // WEST + MIDDLE

  //   env[wrapNorth + wrapEast] + // EAST + NORTH
  //   env[wrapNorth + wrapWest] + // WEST + NORTH

  //   env[wrapSouth + wrapEast] + // EAST + SOUTH
  //   env[wrapSouth + wrapWest] + // WEST + SOUTH

  //   env[wrapNorth + x] + // MIDDLE + TOP
  //   env[wrapSouth + x]; // MIDDLE + BOTTOM

  __syncthreads();

  if(count < 2 || count > 3)
    env[y * size + x] = false;

  if(count == 3)
    env[y * size + x] = true;
}

void print(bool* env) {
  for(int i = 0; i < size * size; i++) {

    cout << (env[i] ? '#' : ' ');

    if (!(i % size)) cout << endl;
  }
}

int main(){

  int parada = 0;

  bool env[size * size];

  // srand(time(NULL));

  // for (int i = 0; i < size * size; i++) {
  //   env[i] = rand() % 2 == 0;
  // }

  env[ 5*size + 7] = true;
  env[ 6*size + 8] = true;
  env[ 8*size +8] = true;
  env[ 6*size +6] = true;
  env[ 8*size +10] = true;
  env[ 9*size +10] = true;
  env[ 8*size +11] = true;
  env[10*size +11] = true;
  env[10*size +12] = true;

  bool* dEnv;

  hipMalloc((void**) &dEnv, size * size * sizeof(bool));
  hipMemcpy(dEnv, env, size * size * sizeof(bool), hipMemcpyHostToDevice);

  dim3 golThreads(size, size);

  while (parada < 100) {
    system("clear");
    jogo<<<1, golThreads>>>(dEnv);
    hipMemcpy(env, dEnv, size * size * sizeof(bool), hipMemcpyDeviceToHost);
    print(env);

    usleep(100000);

    parada++;
  }
}