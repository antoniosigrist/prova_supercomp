
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstring>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define size 21

using namespace std;

__global__ void gpuNext(int* env) {
  int x = threadIdx.x;
  int y = threadIdx.y;

  int wrapNorth = ((size + y - 1) % size) * size;
  int wrapSouth = ((size + y + 1) % size) * size;

  int wrapEast = (size + x + 1) % size;
  int wrapWest = (size + x - 1) % size;

  int neighbours =
    env[y * size + wrapEast] + // EAST + MIDDLE
    env[y * size + wrapWest] + // WEST + MIDDLE

    env[wrapNorth + wrapEast] + // EAST + NORTH
    env[wrapNorth + wrapWest] + // WEST + NORTH

    env[wrapSouth + wrapEast] + // EAST + SOUTH
    env[wrapSouth + wrapWest] + // WEST + SOUTH

    env[wrapNorth + x] + // MIDDLE + TOP
    env[wrapSouth + x]; // MIDDLE + BOTTOM

  __syncthreads();

  if(neighbours < 2 || neighbours > 3)
    env[y * size + x] = 0;

  if(neighbours == 3)
    env[y * size + x] = 1;
}

void print(int* env) {
  for(int i = 0; i < size * size; i++) {
    cout << (env[i] ? '#' : ' ');

    if (!(i % size)) cout << endl;
  }
}

int main(){
  int env[size * size];

  srand(time(NULL));

  for (int i = 0; i < size * size; i++) {
    env[i] = rand() % 2;
  }

  int* dEnv;

  hipMalloc((void**) &dEnv, size * size * sizeof(int));
  hipMemcpy(dEnv, env, size * size * sizeof(int), hipMemcpyHostToDevice);

  dim3 golThreads(size, size);

  while (true) {
    system("clear");
    gpuNext<<<1, golThreads>>>(dEnv);
    hipMemcpy(env, dEnv, size * size * sizeof(int), hipMemcpyDeviceToHost);
    print(env);
    system("sleep .1");
  }
}